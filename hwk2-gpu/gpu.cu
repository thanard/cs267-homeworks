#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256

extern double size;


struct grid
{
    int size;
    linkedlist_t ** grid;
    linkedlist_t ** to_add[8];
};

typedef struct grid grid_t;

//
// initialize grid and fill it with particles
// 
void grid_init(grid_t & grid, grid_t & d_grid, int size)
{
    grid.size = size;

    // Initialize grid
    grid.grid = (linkedlist**) malloc(sizeof(linkedlist*) * size * size);
    memset(grid.grid, 0, sizeof(linkedlist*) * size * size);

    for(int i=0; i<8;i++){
      grid.to_add[i] = (linkedlist**) malloc(sizeof(linkedlist*) * size * size);
      memset(grid.to_add[i], 0, sizeof(linkedlist*) * size * size);
    }
    // if (grid.grid == NULL)
    // {
    //     fprintf(stderr, "Error: Could not allocate memory for the grid!\n");
    //     exit(1);
    // }

    // Set gpu grid
    hipMalloc((void **) &d_grid.grid, size * size * sizeof(linkedlist*));
    for (int i=0; i<8;i++){
      hipMalloc((void **) &d_grid.to_add[i], size * size * sizeof(linkedlist*));
    }
}

//
// adds a particle pointer to the grid
//
void grid_add(grid_t & grid, particle_t * p)
{
    int gridCoord = grid_coord_flat(grid.size, p->x, p->y);

    linkedlist_t * newElement = (linkedlist_t *) malloc(sizeof(linkedlist));
    newElement->value = p;

    // Beginning of critical section
    newElement->next = grid.grid[gridCoord];

    grid.grid[gridCoord] = newElement;
    // End of critical section
}

void grid_copy_to_cuda(grid_t & grid, grid_t & d_grid){
    hipMemcpy(d_grid.grid, grid.grid, size * size * sizeof(linkedlist*), hipMemcpyHostToDevice);
    for (int i=0; i<8;i++){
      hipMemcpy(d_grid.to_add[i], grid.to_add[i], size * size * sizeof(linkedlist*), hipMemcpyHostToDevice);
    }
}
//
// Removes a particle from a grid
//
bool grid_remove(grid_t & grid, particle_t * p, int gridCoord)
{
    if (gridCoord == -1)
        gridCoord = grid_coord_flat(grid.size, p->x, p->y);

    // No elements?
    if (grid.grid[gridCoord] == 0)
    {
        return false;
    }

    // Beginning of critical section

    linkedlist_t ** nodePointer = &(grid.grid[gridCoord]);
    linkedlist_t * current = grid.grid[gridCoord];

    while(current && (current->value != p))
    {
        nodePointer = &(current->next);
        current = current->next;
    }

    if (current)
    {
        *nodePointer = current->next;
        free(current);
    }

    // End of critical section

    return !!current;
}

//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(particle_t * particles, grid_t d_grid, int n)
{
  // Get thread (particle) ID
  int gx = threadIdx.x + blockIdx.x * blockDim.x;
  int gy = threadIdx.y + blockIdx.y * blockDim.y;
  // printf("threadIdx.x=%d, blockIdx.x=%d, blockDim.x=%d\n", threadIdx.x,blockIdx.x, blockDim.x);
  // printf("threadIdx.y=%d, blockIdx.y=%d, blockDim.y=%d\n", threadIdx.y,blockIdx.y, blockDim.y);

  if(gx >= d_grid.size || gy >= d_grid.size) return;

  // particles[tid].ax = particles[tid].ay = 0;
  // for(int j = 0 ; j < n ; j++)
  //   apply_force_gpu(particles[tid], particles[j]);

  // get x y coord of grid from tid.
  // then look at neighbour cells.
  // do apply force.
  linkedlist_t * particle = d_grid.grid[gx*d_grid.size + gy];
  while( particle ){
    for(int x = max(gx - 1, 0); x <= min(gx + 1, d_grid.size-1); x++){
      for(int y = max(gy - 1, 0); y <= min(gy + 1, d_grid.size-1); y++){
        linkedlist_t * curr = d_grid.grid[x * d_grid.size + y];
        while(curr)
          {
              apply_force_gpu(*(particle->value), *(curr->value));
              curr = curr->next;
          }
      }
    }
    particle = particle -> next;
  }
}

// __global__ void move_gpu (particle_t * particles, grid_t d_grid, int n, double size)
// {

//   // Get thread (particle) ID
//   int gx = threadIdx.x + blockIdx.x * blockDim.x;
//   int gy = threadIdx.y + blockIdx.y * blockDim.y;

//   if(gx >= d_grid.size || gy >= d_grid.size) return;

//   //
//   //  slightly simplified Velocity Verlet integration
//   //  conserves energy better than explicit Euler method
//   //
//   p->vx += p->ax * dt;
//   p->vy += p->ay * dt;
//   p->x  += p->vx * dt;
//   p->y  += p->vy * dt;

//   //
//   //  bounce from walls
//   //
//   while( p->x < 0 || p->x > size )
//   {
//       p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
//       p->vx = -(p->vx);
//   }
//   while( p->y < 0 || p->y > size )
//   {
//       p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
//       p->vy = -(p->vy);
//   }

// }



int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    // Set up grids
    int gridSize = (get_size()/get_cutoff()) + 1;
    grid_t grid;
    grid_t d_grid;
    grid_init(grid, d_grid, gridSize);

    for (int i = 0; i < n; ++i)
    {
        grid_add(grid, &particles[i]);
    }
    
    grid_copy_to_cuda(grid, d_grid);
    
    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //

	      // int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
        // int blks = (gridSize * gridSize + NUM_THREADS - 1) / NUM_THREADS;
        dim3 dimBlock((gridSize+15)/16, (gridSize+15)/16, 1);
        dim3 dimThread(16, 16, 1);
        // printf("blk=%d, num_threads=%d\n", blks, NUM_THREADS);
	      compute_forces_gpu <<< dimBlock, dimThread >>> (d_particles, d_grid, n);

        //
        //  move particles
        //
	      // move_gpu <<< dimBlock, dimThread >>> (d_particles, d_grid, n, size);
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	        // Copy the particles back to the CPU
          hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
          save( fsave, n, particles);
	       }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
